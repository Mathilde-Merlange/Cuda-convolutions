#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if( i < cols && j < rows ) {
    g[ j * cols + i ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) / 1024;
  }
}
__global__ void sobel(unsigned char const * const in, unsigned char * const out, std::size_t w, std::size_t h )
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if( i > 1 && i < (w - 1) && j > 1 && j < (h - 1) )
  {
    auto hh = in[ (j-1)*w + i - 1 ] - in[ (j-1)*w + i + 1 ]
           + 2 * in[ j*w + i - 1 ] - 2* in[ j*w+i+1 ]
           + in[ (j+1)*w + i -1] - in[ (j+1)*w +i + 1 ];
    auto vv = in[ (j-1)*w + i - 1 ] - in[ (j+1)*w + i - 1 ]
           + 2 * in[ (j-1)*w + i  ] - 2* in[ (j+1)*w+i ]
           + in[ (j-1)*w + i +1] - in[ (j+1)*w +i + 1 ];

    auto res = hh * hh + vv * vv;
    res = res > 255*255 ? res = 255*255 : res;
    out[ j * w + i ] = sqrt( (float)res );

  }
}

int main()
{
  cv::Mat m_in = cv::imread("../data/in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;
  std::vector< unsigned char > g( rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1, g.data() );
  
  unsigned char * rgb_d;
  unsigned char * g_d;
  
  unsigned char * out_d;
  hipMalloc( &rgb_d, 3 * rows * cols );
  
  hipMalloc( &g_d, rows * cols );
  
  
  hipMalloc( &out_d, rows * cols );
  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  dim3 t(32,32);
  dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );
  
hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );

grayscale<<< b, t >>>( rgb_d, g_d, cols, rows );
  sobel<<< b, t >>>( g_d, out_d, cols, rows );

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  if( err != hipSuccess )
  {
    std::cout << hipGetErrorString( err );
  }

hipMemcpy( g.data(), out_d, rows * cols, hipMemcpyDeviceToHost );

hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";

  cv::imwrite( "graysobel-cu.jpg", m_out );
  hipFree( rgb_d);
  
  hipFree( g_d);
  
  return 0;
}

