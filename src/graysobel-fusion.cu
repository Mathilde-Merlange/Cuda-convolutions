#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void grayscale_sobel( unsigned char * in, unsigned char * out, std::size_t w, std::size_t h ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if( i < w && j < h ) {
    in[ j * w + i ] = (
			 307 * in[ 3 * ( j * w + i ) ]
			 + 604 * in[ 3 * ( j * w + i ) + 1 ]
			 + 113 * in[  3 * ( j * w + i ) + 2 ]
			 ) / 1024;
  }

  __syncthreads();

  if( i > 1 && i < (w - 1) && j > 1 && j < (h - 1) )
  {
    auto hh = in[ (j-1)*w + i - 1 ] - in[ (j-1)*w + i + 1 ]
           + 2 * in[ j*w + i - 1 ] - 2* in[ j*w+i+1 ]
           + in[ (j+1)*w + i -1] - in[ (j+1)*w +i + 1 ];
    auto vv = in[ (j-1)*w + i - 1 ] - in[ (j+1)*w + i - 1 ]
           + 2 * in[ (j-1)*w + i  ] - 2* in[ (j+1)*w+i ]
           + in[ (j-1)*w + i +1] - in[ (j+1)*w +i + 1 ];

    auto res = hh * hh + vv * vv;
    res = res > 255*255 ? res = 255*255 : res;
    out[ j * w + i ] = sqrt( (float)res );

  }
}

int main()
{
  cv::Mat m_in = cv::imread("../data/in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;
  std::vector< unsigned char > g( rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1, g.data() );
  unsigned char * rgb_d;
  unsigned char * g_d;
  unsigned char * out_d;
  hipMalloc( &rgb_d, 3 * rows * cols );
  hipMalloc( &g_d, rows * cols );
  hipMalloc( &out_d, rows * cols );
  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  dim3 t( 32, 32 );
  dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );
  grayscale_sobel<<< b, t >>>( rgb_d, g_d, cols, rows );

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  if( err != hipSuccess )
  {
    std::cout << hipGetErrorString( err );
  }

hipMemcpy( g.data(), g_d, rows * cols, hipMemcpyDeviceToHost );
  cv::imwrite( "fusion.jpg", m_out );
  hipFree( rgb_d);
  hipFree( g_d);
  return 0;
}
